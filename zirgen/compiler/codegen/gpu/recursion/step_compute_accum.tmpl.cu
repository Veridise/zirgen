#include "hip/hip_runtime.h"
// This code is automatically generated

#include "fp.h"
#include "fpext.h"

namespace risc0::circuit::recursion {

__global__ void step_compute_accum(
    const Fp* ctrl, const Fp* data, const Fp* mix, {{pools}}, uint32_t steps, uint32_t count) {
  uint32_t mask = steps - 1;
  uint32_t cycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (cycle >= count) {
    return;
  }
{{#body}}
  {{.}}
{{/body}}
}

} // namespace risc0::circuit::recursion
